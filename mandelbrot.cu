#include "hip/hip_runtime.h"
#include <stdio.h>
#include <X11/Xlib.h>
#include <unistd.h>
#include <stdlib.h>
#include <tiffio.h>
#include <pthread.h>

// =============== STRUCT DEFINITIONS ========================
// the X context
struct drawgc {
     Display *Display;       //     X display connection
     Window window;        //     dialog box
     Pixmap pixmap;    // a Pixmap for later use
     int winx;         // X size
     int winy;         // Y size
     int pixmapx;
     int pixmapy;
     GC gc;            // window Graphics Context
     GC pixmapgc;      // PixMap Graphics Context
     int depth;
} fractwin;

// definition for a specific row in the mandelbrot set
struct row_data{
     double winx;
     double winy;
     double xstart;
     double xscale;
     double xend;
     int max_itterations;
     int complete;
     pthread_t thread;
     double sy;
     unsigned long *colors;
};
// data about the current viewport
struct viewport {
  double viewx;
  double viewy;
  double zoom; 
  double factor;
} fractview;

struct fract_cords {
 int x;
 int y;
} ;


// ===================== HOST X INITIALIZATION =================
int Xinit (){
     char *display_name = NULL;
     XWindowAttributes windowattribs;
     fractwin.Display = XOpenDisplay(display_name);
     if (!fractwin.Display) {
          return 1;
     }
     fractwin.window = XCreateSimpleWindow(fractwin.Display, DefaultRootWindow(fractwin.Display), 10, 10, fractwin.winx,fractwin.winy, 10, 0, 1234);

     fractwin.pixmap = XCreatePixmap(fractwin.Display, DefaultRootWindow(fractwin.Display), fractwin.winx, fractwin.winy, 32);
     XMapWindow(fractwin.Display, fractwin.window);
     XGetWindowAttributes(fractwin.Display, fractwin.window, &windowattribs);
     fractwin.winx=windowattribs.width;
     fractwin.winy=windowattribs.height;
     fractwin.depth=2;
     for (int c=1 ; c < windowattribs.depth ; c++) {
          fractwin.depth *=2;
     }
     fractwin.gc=XCreateGC(fractwin.Display, fractwin.window, 0,0);
     fractwin.pixmapgc=XCreateGC(fractwin.Display, fractwin.pixmap, 0,0);
     XClearWindow(fractwin.Display, fractwin.window);
     XSelectInput(fractwin.Display, fractwin.window, ButtonPressMask|ButtonReleaseMask|KeyPressMask|KeyReleaseMask);
     return 0;  
}




void tiff_write(struct row_data *bitmap, int renderx, int rendery) {
          TIFF *tif=TIFFOpen("test.tif", "w");
          int sampleperpixel=3;
          char * image= (char*)malloc (renderx*rendery*sampleperpixel);
          int index=0;
          for (int y=0 ; y < rendery ; y++) {     
               for (int x=0 ; x < renderx ; x++) {
                    unsigned long i=bitmap[y].colors[x];
                    unsigned long r, g, b;
                    b = i & 255;
                    i = i >> 8;
                    g = i & 255;
                    i = i >> 8;
                    r = i & 255;
                    image[index]=(int)r;
                    index ++;
                    image[index]=(int)g;
                    index ++;
                    image[index]=(int)b;
                    index ++;
               }
          }
          TIFFSetField (tif, TIFFTAG_IMAGEWIDTH, renderx);  // set the width of the image
          TIFFSetField(tif, TIFFTAG_IMAGELENGTH, rendery);    // set the height of the image
          TIFFSetField(tif, TIFFTAG_SAMPLESPERPIXEL, sampleperpixel);   // set number of channels per pixel
          TIFFSetField(tif, TIFFTAG_BITSPERSAMPLE, 8);    // set the size of the channels
          TIFFSetField(tif, TIFFTAG_ORIENTATION, ORIENTATION_TOPLEFT);    // set the origin of the image.
          //   Some other essential fields to set that you do not have to understand for now.
          TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);
          TIFFSetField(tif, TIFFTAG_PHOTOMETRIC, PHOTOMETRIC_RGB);
          tsize_t linebytes = sampleperpixel*renderx;
          unsigned char * buf=(unsigned char*)malloc(linebytes);
          TIFFSetField(tif, TIFFTAG_ROWSPERSTRIP, TIFFDefaultStripSize(tif, sampleperpixel*renderx));
          index=0;
          for (int y=0 ; y < rendery ; y++) {     
               TIFFWriteScanline(tif, &image[index], y, 0);
               index += renderx*sampleperpixel;
          }
          
          TIFFClose(tif);
          free(image);
          free(buf);
}


__global__ void nv_pixel(unsigned long* color, struct row_data *data) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
//    int d_itterations = *max_itterations;
    float sx;
    sx = (index/ data->xend) * data->xscale + data->xstart;
    double ax=0;
    double ay=0;
 
    unsigned long i=0;
    
    while (ax*ax + ay*ay < 4 && i < data->max_itterations) {
         double xtemp = ax*ax - ay*ay + sx;
         ay = 2*ax*ay + data->sy;
         ax = xtemp;
         i++;
    }
    
                 i=i*2;
               unsigned long blue= i & 255;
               i = i >> 8;
               unsigned long green= i & 255;
               i = i >> 8;
               unsigned long red = i & 255;
               i=0;
      //         printf ("%i, %i, %i\n", red, green, blue);
               i=(green << 16) | (blue << 8) | red;

    
    color[index] = i;
}


// test nv kernel ===============================================================================
__global__ void nv_pixel_test(unsigned long* color, struct row_data *data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
//    printf ("In NV Pixel Test for row: %i pixel %i\n", (int) data->winy, index);
    float sx;
    sx = (index/ data->xend) * data->xscale + data->xstart;
    double ax=0;
    double ay=0;
 
    unsigned long i=0;
    while (ax*ax + ay*ay < 4 && i < data->max_itterations) {
         double xtemp = ax*ax - ay*ay + sx;
         ay = 2*ax*ay + data->sy;
         ax = xtemp;
         i++;
    }
    i=i*2;
    unsigned long blue= i & 255;
    i = i >> 8;
    unsigned long green= i & 255;
    i = i >> 8;
    unsigned long red = i & 255;
    i=0;
    i=(green << 16) | (blue << 8) | red;
    __syncthreads();
    color[(int)data->winx * (int)data->winy + index] = i;
//    printf ("In NV Pixel Test for %i, %i i is %i -- %i\n", (int) data->winy, index, (int)i, (int)color[index]);
}

// end test nv kernel ==========================================================================
void *row_thread (void *args) {
     struct row_data *data=(struct row_data*)args;
     for (int x=0 ; x < data->xend ; x++){
          double sx=((x/data->xend)*data->xscale)+data->xstart;
          double ax=0;
          double ay=0;
          unsigned long i=0;
          // per pixel loop
          while (ax*ax + ay*ay < 4 && i < data->max_itterations) {
               double xtemp = ax*ax - ay*ay + sx;
               ay = 2*ax*ay + data->sy;
               ax=xtemp;
               i++;
          }
          data->colors[x]=i;
//          if (i > 255) {
  //             data->colors[x]=0;
  //        } else {
               i=i*2;
               unsigned long blue= i & 255;
               i = i >> 8;
               unsigned long green= i & 255;
               i = i >> 8;
               unsigned long red = i & 255;
               i=0;
      //         printf ("%i, %i, %i\n", red, green, blue);
               i=(green << 16) | (blue << 8) | red;
               data->colors[x]=i;
               
    //      }
//          printf("debug: %f %i %i %i\n", temp, i, data->max_itterations, data->colors[x] );
     }
     data->complete=1;
     return data;

}


//--------------------------
void mandelbrot_threaded(int full, int pixmap) {

     struct timespec starttime;
     struct timespec stoptime;
     int renderx, rendery;
     GC rendergc;
     if (pixmap ==1 ) {
          renderx=fractwin.pixmapx;
          rendery=fractwin.pixmapy;
          rendergc=fractwin.pixmapgc;
     } else {
          renderx=fractwin.winx;
          rendery=fractwin.winy;
          rendergc=fractwin.gc;
     }
     clock_gettime(CLOCK_MONOTONIC, &starttime);
     int max_threads=5;
     // CUDA override
     double max_itterations=100000/fractview.zoom;
     if (!full) {
          max_itterations=100+fractview.factor;
          max_threads=50;
     }
printf("============================================\nCPU Pthread Render with %i Threads\n============================================\n",max_threads);
     printf("Dims: %i, %i Framebuffer %i Bytes\n", renderx, rendery, ((sizeof(struct row_data)*rendery)+(sizeof(unsigned long)*renderx*rendery)));
     int active_threads=0;
     struct row_data *bitmap=(struct row_data*)malloc(sizeof(struct row_data)*rendery);
     double scalex=3.5*fractview.zoom;
     double scaley=2*fractview.zoom;
     printf("Scale: %f, %f Viewport: %f, %f Zoom: %la\n", scalex, scaley, fractview.viewx, fractview.viewy, fractview.zoom);     
     printf("Max Itterations: %f Factor %la \n", max_itterations, fractview.factor);
     for (double y=0 ; y < rendery ; y++){
          double sy=((y/rendery)*scaley)+fractview.viewy;
          bitmap[(int)y].winx=renderx;
          bitmap[(int)y].xstart=fractview.viewx;
          bitmap[(int)y].xscale=scalex;
          bitmap[(int)y].xend=renderx;
          bitmap[(int)y].max_itterations=max_itterations;
          bitmap[(int)y].sy=sy;
          bitmap[(int)y].complete=0;
          bitmap[(int)y].winy=y;
          bitmap[(int)y].colors=(unsigned long*)malloc(sizeof(long)*(bitmap[(int)y].winx+2));
//          printf ("\rspawning thread: %i", (int)y);
          pthread_create(&(bitmap[(int)y].thread),NULL,row_thread,&bitmap[(int)y]);
          active_threads++;
//          printf ("Drawing\n");
          while (active_threads > max_threads) {
                for (int j=0 ; j < (int)y ; j++) {
                     if (bitmap[j].complete==1) {
                          pthread_t thisthread = bitmap[j].thread;
                          struct row_data *returnval=(struct row_data*)malloc(sizeof(struct row_data));
                          pthread_join(thisthread, (void**)returnval);
                          bitmap[j].complete=2;
                          active_threads--;
                          for (int k=0 ; k < renderx ; k++) {
                               XSetForeground (fractwin.Display, rendergc, bitmap[j].colors[k]);
                               if (pixmap ==0) {
                                    XDrawPoint(fractwin.Display, fractwin.window, rendergc, k, bitmap[j].winy);
                               } else {
                                    XDrawPoint(fractwin.Display, fractwin.pixmap, rendergc, k, bitmap[j].winy);
                               }
                          } // draw rows

                     } // row complete
                     
                } // itterate through launched rows
                XFlush(fractwin.Display);
          } // if we have exceeded max_threads
     } // Y loop
    for (int y=0 ; y < rendery ; y++) {     
          if (bitmap[y].complete==1) {
               pthread_t thisthread = bitmap[y].thread;
               struct row_data *returnval=(struct row_data*)malloc(sizeof(struct row_data));
               pthread_join(thisthread, (void**)returnval);
               bitmap[y].complete=2;
          }
          for (int x=0 ; x < renderx ; x++) {
               XSetForeground (fractwin.Display, fractwin.gc, bitmap[y].colors[x]);
               if (pixmap==0) {
                    XDrawPoint(fractwin.Display, fractwin.window, fractwin.gc, x, bitmap[y].winy);
               } else {
                    XDrawPoint(fractwin.Display, fractwin.pixmap, fractwin.pixmapgc, x, bitmap[y].winy);
               }
               } // draw rows
     } // ensure everything is drawn
     XFlush(fractwin.Display);
     clock_gettime(CLOCK_MONOTONIC, &stoptime);
     printf("\ncompleted in %ld seconds.\n", (stoptime.tv_sec - starttime.tv_sec));
          if (pixmap==1) {
//          XWriteBitmapFile(fractwin.Display, "test.xbm", fractwin.pixmap, renderx, rendery, -1, -1);
          tiff_write(bitmap, renderx, rendery);
     }
     for (int y=0 ; y < (int)rendery ; y++) {
	free(bitmap[y].colors);
	}
     free (bitmap);    
     
printf("============================================\nThread Render Complete\n============================================\n");
}

//---------------------------
void mandelbrot_cuda(int full, int pixmap) {
// The Mandlebrot set is interesting in the real region x = -2 to +1 and y= -1 to +1.

     struct timespec starttime;
     struct timespec stoptime;
     int renderx, rendery;
     double max_itterations;
     GC rendergc;
     if (pixmap ==1 ) {
          renderx=fractwin.pixmapx;
          rendery=fractwin.pixmapy;
          rendergc=fractwin.pixmapgc;
     } else {
          renderx=fractwin.winx;
          rendery=fractwin.winy;
          rendergc=fractwin.gc;
     }
     clock_gettime(CLOCK_MONOTONIC, &starttime);
     max_itterations=100000/fractview.zoom;
     if (!full) {
          max_itterations=100+fractview.factor;
     }

     
     struct row_data *bitmap= (struct row_data*)malloc(sizeof(struct row_data)*rendery);
     double scalex=3.5*fractview.zoom;
     double scaley=2*fractview.zoom;

     struct row_data * d_row;
     unsigned long *d_color;
     
     int threadcount = 512; //multiples of 32 max 1024 per Nvidia documentation
          //  smallest executable unit of parallelism on a CUDA device comprises 32 threads
          // http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html
     int blockcount=((renderx + threadcount - 1)/threadcount);
     printf("============================================\nCUDA Render with %i blocks of %i threads\n============================================\n", blockcount, threadcount);
     printf("Dims: %i, %i Framebuffer %i Bytes\n", renderx, rendery, ((sizeof(struct row_data)*rendery)+(sizeof(unsigned long)*renderx*rendery)));
     printf("Scale: %f, %f Viewport: %f, %f Zoom: %la\n", scalex, scaley, fractview.viewx, fractview.viewy, fractview.zoom);
     printf("Max Itterations: %f Factor %la \n", max_itterations, fractview.factor);
     size_t mysize = sizeof(long) * renderx;
     hipMalloc ((void**)&d_row, sizeof(struct row_data));
     hipMalloc ((void**)&d_color, mysize);
     for (double y=0 ; y < rendery ; y++) {
          double sy=((y/rendery)*scaley)+fractview.viewy;
          bitmap[(int)y].winx=renderx;
          bitmap[(int)y].xstart=fractview.viewx;
          bitmap[(int)y].xscale=scalex;
          bitmap[(int)y].xend=renderx;
          bitmap[(int)y].max_itterations=max_itterations;
          bitmap[(int)y].sy=sy;
          bitmap[(int)y].winy=y;
          bitmap[(int)y].colors=(unsigned long*)malloc(sizeof(unsigned long)*(bitmap[(int)y].winx+2));
          
          //----------------- CUDA_ROW -----------
          
          hipMemcpy(d_row, &bitmap[(int)y], sizeof(struct row_data), hipMemcpyHostToDevice);
//          hipMemcpy(d_color, bitmap[(int)y].colors, mysize, hipMemcpyHostToDevice);

          nv_pixel<<<blockcount,threadcount>>>(d_color, d_row);
          
          // draw previous row
          if (y > 0) {
               int k;
               for (k=0 ; k < fractwin.winx ; k++) {
                    XSetForeground (fractwin.Display, fractwin.gc, bitmap[(int)y-1].colors[k]);
                    if (pixmap == 0) {
                         XDrawPoint(fractwin.Display, fractwin.window, rendergc, k, bitmap[(int)y-1].winy);
                    } else {
                         XDrawPoint(fractwin.Display, fractwin.pixmap, rendergc, k, bitmap[(int)y-1].winy);
                    }
                }
                XFlush(fractwin.Display);
           }
           // wait for Cuda
           
//          hipDeviceSynchronize();
          hipMemcpy(bitmap[(int)y].colors,d_color, mysize, hipMemcpyDeviceToHost);
          
          //------------------------------------
          
          
     } // main render loop
     hipFree(d_row); 
     hipFree(d_color);
     hipDeviceSynchronize();
     for (int y=0 ; y < rendery ; y++) {
          for (int x=0 ; x < renderx ; x++) {
               XSetForeground (fractwin.Display, fractwin.gc, bitmap[y].colors[x]);
               if (pixmap==0) {
                    XDrawPoint(fractwin.Display, fractwin.window, fractwin.gc, x, bitmap[y].winy);
               } else {
                    XDrawPoint(fractwin.Display, fractwin.pixmap, fractwin.pixmapgc, x, bitmap[y].winy);
               }
          } // draw rows
     } // ensure everything is drawn
     XFlush(fractwin.Display);
     clock_gettime(CLOCK_MONOTONIC, &stoptime);
     printf("\ncompleted in %ld seconds.\n", (stoptime.tv_sec - starttime.tv_sec));
//     printf("factor is: %lf, %lf\n", fractview.factor, fractview.zoom);
     if (pixmap==1) {
          tiff_write(bitmap, renderx, rendery);
     }
     for (int y=0 ; y < (int)rendery ; y++) {
	free(bitmap[y].colors);
	}
     free (bitmap); 
     printf("============================================\nCUDA Render Complete\n============================================\n");  
}


// test routine =====================================================================================================================

void mandelbrot_test(int full, int pixmap) {
// The Mandlebrot set is interesting in the real region x = -2 to +1 and y= -1 to +1.
     struct timespec starttime;
     struct timespec stoptime;
     int renderx, rendery;
     double max_itterations;
     GC rendergc;
     if (pixmap ==1 ) {
          renderx=fractwin.pixmapx;
          rendery=fractwin.pixmapy;
          rendergc=fractwin.pixmapgc;
     } else {
          renderx=fractwin.winx;
          rendery=fractwin.winy;
          rendergc=fractwin.gc;
     }
     clock_gettime(CLOCK_MONOTONIC, &starttime);
     max_itterations=100000/fractview.zoom;
     if (!full) {
          max_itterations=1000+fractview.factor;
     }
     
     struct row_data *bitmap= (struct row_data*)malloc(sizeof(struct row_data)*rendery);
     double scalex=3.5*fractview.zoom;
     double scaley=2*fractview.zoom;

     struct row_data *d_row;
     unsigned long *d_color;
  //   unsigned long  *h_color;
     
     int threadcount = 64; //multiples of 32 max 1024 per Nvidia documentation
          //  smallest executable unit of parallelism on a CUDA device comprises 32 threads
          // http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html
     int blockcount=((renderx + threadcount - 1)/threadcount);
     printf("============================================\nCUDA Test Render with %i blocks of %i threads\n============================================\n", blockcount, threadcount);
     
     printf("Dims: %i, %i Framebuffer %i Bytes\n", renderx, rendery, (int)(rendery * renderx * sizeof(unsigned long)));
     printf("Scale: %f, %f Viewport: %f, %f Zoom: %l\n", scalex, scaley, fractview.viewx, fractview.viewy, fractview.zoom);
     printf("Max Itterations: %f Factor %l \n", max_itterations, fractview.factor);
     
     // initialization 
     
     hipStream_t *streams = (hipStream_t *)malloc(sizeof(hipStream_t) * rendery);
     hipMalloc ((void**)&d_row, (sizeof(struct row_data)*rendery));
 //    hipMalloc ((void**)&d_color, renderx * sizeof(unsigned long));
//     hipHostMalloc ((void**)&h_color, (sizeof(unsigned long)*rendery*renderx));
     
     // collect input
	printf ("memory definition\n");
     for (double y=0 ; y < rendery ; y++) {
          double sy=((y/rendery)*scaley)+fractview.viewy;
          bitmap[(int)y].winx=renderx;
          bitmap[(int)y].xstart=fractview.viewx;
          bitmap[(int)y].xscale=scalex;
          bitmap[(int)y].xend=renderx;
          bitmap[(int)y].max_itterations=max_itterations;
          bitmap[(int)y].sy=sy;
          bitmap[(int)y].winy=y;
	  bitmap[(int)y].complete=0;
          bitmap[(int)y].colors=(unsigned long*)malloc(sizeof(unsigned long)*(bitmap[(int)y].winx));
     }
     
     // send to CUDA
     hipMemcpy((void*)d_row, (void*)bitmap, (sizeof(struct row_data)*rendery), hipMemcpyHostToDevice);
printf ("Spawning Kernels \n");
int running = 0;
     hipMalloc (&d_color, rendery * renderx * sizeof(unsigned long));
     for (int y=0 ; y < (int)rendery ; y++) {
          printf ("\rSpawning Kernel %i ", y);
          if (running < 100 ) {
          	hipStreamCreate(&streams[y]);
	          nv_pixel_test<<<blockcount,threadcount,0, streams[y]>>>(d_color, &d_row[y]);
		running++;
          } else {
		for (int t=0 ; t < y ; t++) {
			if (!bitmap[t].complete && (hipStreamQuery(streams[t])==hipSuccess)) {
				bitmap[t].complete=1;
				hipMemcpyAsync(bitmap[t].colors, &d_color[t*(int)renderx], sizeof (unsigned long)*renderx, hipMemcpyDeviceToHost, streams[t]);
				hipStreamDestroy(streams[t]);
				running--;
                                printf ("\rDrawing Row %i",t);
				for (int x=0 ; x < renderx ; x++) {
			               XSetForeground (fractwin.Display, fractwin.gc, bitmap[t].colors[x]); // working
			               if (pixmap == 0) {
                        			 XDrawPoint(fractwin.Display, fractwin.window, rendergc, x, t);
			               } else {
                        			 XDrawPoint(fractwin.Display, fractwin.pixmap, rendergc, x, t);
			               }
			          }
			}
                }
		y--;
          }
          
     }
     
     printf ("\rcleaning up       \n");
     // get returns and render
     for (int y=0 ; y < (int)rendery ; y++) {
          
//          printf ("\rCompleting Cuda Row %i.",y);
//          hipStreamSynchronize(streams[y]);   
          if (bitmap[y].complete==0) {     
	        hipMemcpyAsync(bitmap[y].colors, &d_color[y*(int)renderx], sizeof (unsigned long)*renderx, hipMemcpyDeviceToHost, streams[y]); // working
          	hipStreamDestroy(streams[y]);
		printf ("\rDrawing Row %i",y);
		for (int x=0 ; x < renderx ; x++) {
               XSetForeground (fractwin.Display, fractwin.gc, bitmap[y].colors[x]); // working
               if (pixmap == 0) {
                         XDrawPoint(fractwin.Display, fractwin.window, rendergc, x, y);
                    } else {
                         XDrawPoint(fractwin.Display, fractwin.pixmap, rendergc, x, y);
                    }
          }
	  }
     }

	hipFree(d_color);
	hipFree(d_row);
        free (streams);
// refreshing screen

     XFlush(fractwin.Display);
     clock_gettime(CLOCK_MONOTONIC, &stoptime);
     printf("\ncompleted in %ld seconds.\n", (stoptime.tv_sec - starttime.tv_sec));
     if (pixmap==1) {
          tiff_write(bitmap, renderx, rendery);
     }
     for (int y=0 ; y < (int)rendery ; y++) {
	free(bitmap[y].colors);
	}
     free (bitmap); 
     printf("============================================\nCUDA Test Render Complete\n============================================\n");  
}

// end test routine =================================================================================================================




void zoom (int direction) {
//double zoomnew;
     if (direction == 0) { 	// zoom out
          fractview.zoom += fractview.factor/1000;
          if (fractview.zoom > (fractview.factor/1000)) {
               fractview.factor *= 10;
          }
          if (fractview.factor > 10) {
               fractview.factor = 10;
          }
//         fractview.factor += .1/fractview.factor;
//        fractview.factor += 1;
     } else {			// zoom in
//          zoomnew = fractview.zoom - (fractview.factor/1000);
//          if (zoomnew < 0) {
//          fractview.factor -= 1/fractview.factor;
//          }
          if (fractview.zoom - (fractview.factor/1000) <=0) {
               fractview.factor /= 10;
          }
          fractview.zoom -= fractview.factor/1000;
//          fractview.factor -= 1/fractview.factor;
//          fractview.factor -= 1;
          if (fractview.zoom < 0 ) {
//               fractview.zoom=1;
//               fractview.factor=10;
          }
     }
}

struct fract_cords start;
     struct fract_cords end;

int main (int argc, char* argv[]) {
// initialization 
     fractwin.winx=1024;
     fractwin.winy=768;
     if (argc == 3) {
          fractwin.winx=atoi(argv[1]);
          fractwin.winy=atoi(argv[2]);
          
     }
     fractwin.pixmapx=fractwin.winx;
     fractwin.pixmapy=fractwin.winy;
      if (fractwin.winx > 1920) {
          fractwin.winx=1920;
     }
     if (fractwin.winy > 1080) {
          fractwin.winy=1080;
     }

     fractview.viewx = -2;
     fractview.viewy = -1;
     fractview.zoom=1;
     fractview.factor=10;

// initialize X
     Xinit();

     

     int quit=1;
     float scalex;
     float scaley;
     while (quit) {
          Window root_return, child_return;
          int root_x_return, root_y_return;
          unsigned int mask_return;
          int mouse_x, mouse_y;
          XEvent event;
          XNextEvent(fractwin.Display, &event);
/*
q -- Exit
c -- zoom in
d -- zoom out
r -- cuda single xwin
f -- cuda single tiff
y -- test xwin
h -- test tiff
t -- cpu xwin
g -- cpu tiff
' ' -- preview render
right click -- cuda xwin
middle click -- cuda tiff
left click -- preview render
scroll -- zoom
*/
          switch (event.type) {
               case (ButtonPress): {
//                    printf("Buttonpress %i\n", event.xbutton.button);
                    switch(event.xbutton.button) {
                         case (1): { // left click
                              scalex=3.5*fractview.zoom;
                              scaley=2*fractview.zoom;
                              XQueryPointer(fractwin.Display, fractwin.window, &child_return, &root_return, &root_x_return, &root_y_return, &mouse_x, &mouse_y, &mask_return);
                              start.y=mouse_y;
                              start.x=mouse_x;
 //                             printf("\n--------------------\n\nStart is: %i, %i\n\n\n", start.x, start.y);
                              break;
                         }
                         
                    }
                    break;
               } // ButtonPress
               case (ButtonRelease): {
 //                   printf("Buttonrelease %i\n", event.xbutton.button);
                    switch(event.xbutton.button) {
                        case(1): // left click
                        {
                              scalex=3.5*fractview.zoom;
                              scaley=2*fractview.zoom;
                              XQueryPointer(fractwin.Display, fractwin.window, &child_return, &root_return, &root_x_return, &root_y_return, &mouse_x, &mouse_y, &mask_return);
                              end.y=mouse_y;
                              end.x=mouse_x;
                              int deltax = start.x-end.x;
                              int deltay = start.y-end.y;
                              float xtemp=((float)deltax/(float)fractwin.winx)*scalex;
                              float ytemp=((float)deltay/(float)fractwin.winy)*scaley;
//                              printf ("------\ndebug output:  \n%i, %i, %f\n-------\n", deltay, fractwin.winy, ((float)deltay/(float)fractwin.winy));
//                              printf("translation: %f, %f, %i \n", deltay, ytemp, fractwin.winy);
                              fractview.viewx += xtemp;
                              fractview.viewy += ytemp;
//                              printf("\n\n\nDelta: %i, %i to %i, %i\n", start.x, start.y, end.x, end.y);
//                              printf("Delta: %i, %i \n", deltax, deltay);
                              
//                              printf("Viewport moving to: %f, %f \n", fractview.viewx, fractview.viewy);
                              if (fractview.viewy > 1) {
                                   fractview.viewy = 1;
                              }
                              if (fractview.viewy < -1) {
                                   fractview.viewy = -1;
                              }
                              if (fractview.viewx > 1 ) {
                                   fractview.viewx = 1;
                              }
                              if (fractview.viewx < -2) {
                                   fractview.viewx = -2;
                              }
                              if ((start.x != end.x) || (start.y != end.y)) {
                              start.x=0;
                              start.y=0;
                              }
                              end.x=0;
                              end.y=0;
                              if (XPending(fractwin.Display)==0) {
                                   mandelbrot_cuda(0,0);
                              }
                              break;           
                         }
                         case (2): // middle click
                         {
                              mandelbrot_cuda(1,1);
                         }
                         case(3): // right click
                         {
                              mandelbrot_cuda(1,0);
                              break;
                         }
                         case(4): // scroll up
                         {
                              zoom(0);
                              if (XPending(fractwin.Display)==0) {
                                   mandelbrot_cuda(0,0);
                              }
                              break;
                         }
                         case(5): // scroll down
                         {
                              zoom(1);
                              if (XPending(fractwin.Display)==0) {
                                   mandelbrot_cuda(0,0);
                              }
                              break;
                         } 
                    } // switch button
                    break;
               } // ButtonRelease
          
               case (KeyPress): {
//                    printf("Keypress %i\n", event.xbutton.button);
                    switch(event.xkey.keycode) {
                         case (40):
                         {
                              zoom(0);
                              if (XPending(fractwin.Display)==0) {
                                   mandelbrot_cuda(0,0);
                              }
                              break;
                         }
                         case (54):
                         {
                              zoom(1);
                              if (XPending(fractwin.Display)==0) {
                                   mandelbrot_cuda(0,0);
                              }
                              break;
                         }

                    } // keycode
                    break;
               } // KeyPress
                    
               case (KeyRelease): {
                    printf("Keyrelease %i\n", event.xbutton.button);
                    switch (event.xkey.keycode) {
                         case(24):
                         {
                              quit=0;
                              break;
                         }
			case(27):
                         {
                              mandelbrot_cuda(1,0);
                              break;
                         }
			case(41):
                         {
                              mandelbrot_cuda(1,1);
				break;
                         }
                         case(28):
                         {
                              mandelbrot_threaded(1,0);
                              break;
                         }
			case(42):
                         {
                              mandelbrot_threaded(1,1);
				break;
                         }
                         case(29):
                         {
                              mandelbrot_test(1,0);
				break;
                         }
			case(43):
                         {
                              mandelbrot_test(1,1);
				break;
                         }
			 case(65):
                         {
                              mandelbrot_test(0,0);
				break;
                         }

                    } // keycode
                    break;
               } // KeyRelease
          } // switch event type      
     } // main event loop

// normal exit
     return 0;
}